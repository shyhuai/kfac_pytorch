#include <torch/extension.h>
#include <ATen/ATen.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipblas.h>

#include <helper_functions.h>
#include <hip/hip_runtime_api.h>
#include <vector>

#include "tcmm_kernel.h"

using namespace std;

static hipsolverHandle_t g_cusolverH = NULL;
static hipblasHandle_t g_cublasHandle = NULL;
static hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;


#define checkKernelErrors(expr) do {                                                        \
    expr;                                                                                   \
                                                                                            \
    hipError_t __err = hipGetLastError();                                                 \
    if (__err != hipSuccess) {                                                             \
        printf("Line %d: '%s' failed: %s\n", __LINE__, # expr, hipGetErrorString(__err));  \
        abort();                                                                            \
    }                                                                                       \
} while(0)


#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

hipsolverHandle_t get_cusolver_handler() {
    if (g_cusolverH == NULL) {
        cusolver_status = hipsolverDnCreate(&g_cusolverH);
        assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    }
    return g_cusolverH;
}

hipblasHandle_t get_cublas_handler() {
    if (g_cublasHandle == NULL) {
        cublasErrCheck(hipblasCreate(&g_cublasHandle));
    }
    return g_cublasHandle;
}


std::vector<torch::Tensor> tcmm_symeig(torch::Tensor a) {
    const auto a_shape = a.sizes();
    const int m = a_shape[0];
    const int lda = m;
    int lwork = 0;
    int *devInfo = NULL;
    hipError_t cudaStat1 = hipSuccess;

    auto options_float =
        torch::TensorOptions()
        .dtype(a.dtype())
        .layout(torch::kStrided)
        .device(a.device().type())
        .requires_grad(false);

    auto A = a.data_ptr<float>();
    auto V = torch::zeros({m, m}, options_float).copy_(a); // eigenvectors
    //auto V = a.copy_(a);
    auto W = torch::zeros({m}, options_float); // eigenvalues

    hipsolverHandle_t cusolverH = get_cusolver_handler();
    hipsolverEigMode_t jobz = HIPSOLVER_EIG_MODE_VECTOR; // compute eigenvalues and eigenvectors.
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_UPPER;

    cusolver_status = hipsolverDnSsyevd_bufferSize(
            cusolverH,
            jobz,
            uplo,
            m,
            V.data_ptr<float>(),
            lda,
            W.data_ptr<float>(),
            &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    float *d_work = NULL;
    cudaStat1 = hipMalloc((void**)&d_work, sizeof(float)*lwork);
    assert(hipSuccess == cudaStat1);
    cudaStat1 = hipMalloc ((void**)&devInfo, sizeof(int));
    assert(hipSuccess == cudaStat1);

    cusolver_status = hipsolverDnSsyevd(
            cusolverH,
            jobz,
            uplo,
            m,
            V.data_ptr<float>(),
            lda,
            W.data_ptr<float>(),
            d_work,
            lwork,
            devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    hipFree(devInfo);
    hipFree(d_work);
    std::vector<torch::Tensor> tuple;
    tuple.push_back(W); 
    tuple.push_back(V); 
    return tuple;
}

std::vector<torch::Tensor> tcmm_symeig_sparse(torch::Tensor a) {
    std::vector<torch::Tensor> tuple;
    tuple.push_back(a); 
    tuple.push_back(a); 
    return tuple;
}

torch::Tensor tcmm_gemm_ex(torch::Tensor a, torch::Tensor b) {
    torch::Tensor a_fp16 = at::_cast_Half(a);
    torch::Tensor b_fp16 = at::_cast_Half(b);

    const auto a_shape = a.sizes();
    const int m = a_shape[0];
    const int k = a_shape[1];
    const int n = b.sizes()[1];
    const float alpha = 1.0;
    const float beta = 0.0;
    hipError_t cudaStat1 = hipSuccess;

    auto options_float =
        torch::TensorOptions()
        .dtype(torch::kFloat32)
        .layout(torch::kStrided)
        .device(a.device().type())
        .requires_grad(false);
    auto c = torch::zeros({m, n}, options_float); 
    hipblasHandle_t cublasHandle = get_cublas_handler();

    hipblasGemmEx(cublasHandle, HIPBLAS_OP_T, HIPBLAS_OP_T, 
                m, n, k, 
                &alpha,
                a_fp16.data_ptr<at::Half>(), HIP_R_16F, k,
                b_fp16.data_ptr<at::Half>(), HIP_R_16F, n,
                &beta, 
                c.data_ptr<float>(), HIP_R_32F, n,
                HIP_R_32F, CUBLAS_GEMM_DFALT_TENSOR_OP);
    cudaStat1 = hipDeviceSynchronize();
    assert(hipSuccess == cudaStat1);
    return c;
}
